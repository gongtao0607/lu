
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <cmath>
#include <cstdlib>
#include <sys/timeb.h>
#define diffftime(a,b) ((a.time-b.time)+(a.millitm-b.millitm)/1000.0)

//Fix some platforms missing barrier implementation
#if defined(__APPLE__) || defined(__CYGWIN32__) || defined(__CYGWIN64__)
#include "pthread_barrier.h"
#endif

//define BARRIER() and CUDA_CALLABLE differently
#ifdef __HIPCC__ 
#define CUDA_CALLABLE __host__ __device__
#define BARRIER() __syncthreads()
#else
#define CUDA_CALLABLE
#include <pthread.h>
#define BARRIER() pthread_barrier_wait(&barrier)
#endif 

#ifdef GRAPHITE
#include "carbon_user.h"
#endif

//Fix VC missing rand48 functions
#if defined(_MSC_VER)
#define lrand48() rand()
#define srand48(x) srand(x)
#endif 

using namespace std;
int N,P,B,NB;
struct timeb main_t1, main_t2, threads_t1, threads_t2;
class Block{
public:
	float*p;
	int b;
	Block(){p=NULL;}
	Block(int _b){p=NULL;b=_b;}
	CUDA_CALLABLE float& at(int x,int y){
		return p[x*b+y];
	}
};
class Matrix{
public:
	Block*p;
	int b;
	int nb;
	Matrix(){p=NULL;}
	Matrix(int _n,int _b){p=NULL;nb=_n/_b;b=_b;}
	CUDA_CALLABLE Block& getBlock(int x,int y){
		return p[x*nb+y];
	}
	CUDA_CALLABLE float& at(int x,int y){
		return getBlock(x/b,y/b).at(x%b,y%b);
	}
	void print(){
		for(int i=0;i<N;++i){
			for(int j=0;j<N;++j){
				cout<<at(i,j)<<" ";
			}
			cout<<endl;
		}
	}
}*A;
#ifndef __CUDACC__
pthread_barrier_t barrier;
#endif
void usage(char*n){
	cout<<"Usage: ./"<<n<<" N P B matrix_file\n\tN:NxN matrix\n\tP:P threads\n\tB:BxB block"<<endl;
}

// block b, referce block r, line "i" in b, line "k" in r, start column j, alpha
CUDA_CALLABLE void daxpy(Block&b , Block&r, int i, int k, int j, float alpha)
{
	for (int p = j; p<b.b; p++)     b.at(i,p) += alpha*r.at(k,p);
}
CUDA_CALLABLE void top_left(Block&b)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		/* modify subsequent columns */
		for (int i=k+1; i<b.b; i++) {
			b.at(i,k)/= b.at(k,k);
			alpha = -b.at(i,k);
			//length = n-k-1;
			daxpy(b, b, i, k, k+1, alpha);
		}
	}
}


CUDA_CALLABLE void top_right(Block&b, Block&r)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		for (int i=k+1; i<b.b; i++) {
			alpha = -r.at(i,k);
			daxpy(b, b, i, k, 0, alpha);
		}
	}
}


CUDA_CALLABLE void bottom_left(Block &b, Block&r)
{
	float alpha;
	for (int k=0; k<b.b; k++)
		for (int i=0; i<b.b; i++) {
			b.at(i,k) /= r.at(k,k);
			alpha = -b.at(i,k);
			daxpy(b, r, i, k, k+1, alpha);
		}
}


CUDA_CALLABLE void bottom_right(Block &b, Block&r1, Block&r2)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		for (int i=0; i<b.b; i++) {
			alpha = -r1.at(i,k);
			daxpy(b,r2,i,k,0,alpha);
		}
	}
}

#ifdef __HIPCC__
__global__ void thread_main(Matrix*A,int P)
#else
void* thread_main(void*p)
#endif
{

#ifdef __HIPCC__
	int thread_id=threadIdx.x;
#else
	int thread_id=*(int*)p;
	
#endif
	for(int round=0;round<A->nb;++round){
		//step 1, calculate top left
		if(thread_id==0){
			//A->getBlock(round,round);
			top_left(A->getBlock(round,round));
		}
		BARRIER();
		//step 2, bottom left and top right
		//x 0 2
		//1 x x
		//3 x x
		//0,1,2,3->()
		for(int i=thread_id;i<(A->nb-round-1)*2;i+=P){
			if(i&1){
				//bottom left
				//A->getBlock(round+(i>>1)+1,round);
				bottom_left(A->getBlock(round+(i>>1)+1,round),A->getBlock(round,round));
			}else{
				//top right
				//A->getBlock(round,round+(i>>1)+1);
				top_right(A->getBlock(round,round+(i>>1)+1),A->getBlock(round,round));
			}
		}
		BARRIER();
		//step 3, bottom right
		//x x x
		//x 0 1
		//x 2 3
		for(int i=thread_id;i<(A->nb-round-1)*(A->nb-round-1);i+=P){
			//A->getBlock(i/(NB-round-1)+round+1,i%(NB-round-1)+round+1)
			bottom_right(A->getBlock(i/(A->nb-round-1)+round+1,i%(A->nb-round-1)+round+1),
				A->getBlock(i/(A->nb-round-1)+round+1,round),
				A->getBlock(round,i%(A->nb-round-1)+round+1));
		}
		BARRIER();
	}
#ifndef __CUDACC__
	return NULL;
#endif
}

#ifdef __HIPCC__
void cuda_upload(Matrix*rm, Matrix*lm)
{
	Block*blocks=new Block[NB*NB];
#ifdef CONTIGUOUS
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			float*p;
			if(cudaMalloc((void **)&p, sizeof(float)*B*B)!=cudaSuccess)
				throw runtime_error("cudaMalloc");
			blocks[i*NB+j].b=B;
			blocks[i*NB+j].p=p;
			cudaMemcpy(p, lm->getBlock(i,j).p, sizeof(float)*B*B, cudaMemcpyHostToDevice);
		}
	}
#else
	float*p;
	if(hipMalloc((void **)&p, sizeof(float)*B*B*NB*NB)!=hipSuccess)
		throw runtime_error("hipMalloc");
	hipMemcpy(p, lm->getBlock(0,0).p, sizeof(float)*B*B*NB*NB, hipMemcpyHostToDevice);
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			blocks[i*NB+j].b=B;
			blocks[i*NB+j].p=p+B*B*(i*NB+j);
		}
	}
#endif
	Block*d_blocks;
	if(hipMalloc((void **)&d_blocks, sizeof(Block)*NB*NB)!=hipSuccess)
		throw runtime_error("hipMalloc");
	hipMemcpy(d_blocks, blocks, sizeof(Block)*NB*NB, hipMemcpyHostToDevice);
	
	Matrix*m=new Matrix(N,B);
	m->p=d_blocks;
	hipMemcpy(rm, m, sizeof(Matrix), hipMemcpyHostToDevice);
	
	delete m;
	delete[]blocks;
}
void cuda_download(Matrix*lm, Matrix*rm)
{
	hipMemcpy(lm, rm, sizeof(Matrix), hipMemcpyDeviceToHost);
	
	Block*blocks=new Block[NB*NB];
	hipMemcpy(blocks, lm->p, sizeof(Block)*NB*NB, hipMemcpyDeviceToHost);
	hipFree(lm->p);
	
	lm->p=blocks;
#ifdef CONTIGUOUS
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			float*p=new float[B*B];
			cudaMemcpy(p, lm->getBlock(i,j).p, sizeof(float)*B*B, cudaMemcpyDeviceToHost);
			cudaFree(lm->getBlock(i,j).p);
			lm->getBlock(i,j).p=p;
		}
	}
#else
	float*p=new float[B*B*NB*NB];
	hipMemcpy(p, lm->getBlock(0,0).p, sizeof(float)*B*B*NB*NB, hipMemcpyDeviceToHost);
	hipFree(lm->getBlock(0,0).p);
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			lm->getBlock(i,j).p=p+B*B*(i*NB+j);
		}
	}
#endif
}
#endif

int main(int argc, char**argv)
{
	if(argc<=3){
		usage(argv[0]);
		return -1;
	}
	N=atoi(argv[1]);
	P=atoi(argv[2]);
	B=atoi(argv[3]);
	if(N==0||P==0||B==0){
		usage(argv[0]);
		return -1;
	}
	ftime(&main_t1);
	NB=N/B;
	A=new Matrix(N,B);
	A->p=new Block[NB*NB];
	int i,j;
#ifdef CONTIGUOUS
	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			A->getBlock(i,j).b=B;
			A->getBlock(i,j).p=new float[B*B];
		}
	}
#else
	float*p=new float[B*B*NB*NB];
	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			A->getBlock(i,j).b=B;
			A->getBlock(i,j).p=p+B*B*(i*NB+j);
		}
	}
#endif
	srand48(1);
#define MAXRAND 32768.0
	for(i=0;i<N;++i){
		for(j=0;j<N;++j){
			A->at(i,j)=((double)lrand48())/MAXRAND;
			if(i==j){
				A->at(i,j)*=10;
			}
		}
	}

#ifdef __HIPCC__
	//CUDA code begins
	Matrix*d_A;
	if(hipMalloc((void **)&d_A, sizeof(Matrix))!=hipSuccess)
		throw runtime_error("hipMalloc");
	cuda_upload(d_A,A);

	ftime(&threads_t1);
	thread_main<<<1,P>>>(d_A,P);
	hipDeviceSynchronize();
	ftime(&threads_t2);

#ifdef CONTIGUOUS
	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			delete[] A->getBlock(i,j).p;
		}
	}
#else
	delete[] A->getBlock(0,0).p;
#endif
	delete[]A->p;

	cuda_download(A,d_A);
	//CUDA code ends
#else
	//pthread code begins
#ifdef GRAPHITE
	CarbonEnableModels();
#endif
	pthread_barrier_init(&barrier,NULL,P);
	int*thread_args=new int[P];
	pthread_t*thread_handle=new pthread_t[P];
	ftime(&threads_t1);
	for(i=1;i<P;++i){
		thread_args[i]=i;
		pthread_create(&thread_handle[i],NULL,thread_main,&thread_args[i]);
	}
	thread_args[0]=0;
	thread_main(&thread_args[0]);
	for(i=1;i<P;++i){
		pthread_join(thread_handle[i],NULL);
	}
	ftime(&threads_t2);
	delete[]thread_args;
	delete[]thread_handle;
	pthread_barrier_destroy(&barrier);
#ifdef GRAPHITE
	CarbonDisableModels();
#endif
	//pthread code ends
#endif
#ifndef GRAPHITE
	//A->print();
#endif

#ifdef CONTIGUOUS
	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			delete[]A->getBlock(i,j).p;
		}
	}
#else
	delete[]A->getBlock(0,0).p;
#endif
	delete[]A->p;
	delete A;
	ftime(&main_t2);
	cout<<"Overall execution time = "<<diffftime(main_t2,main_t1)<<endl;
	cout<<"Threads execution time = "<<diffftime(threads_t2,threads_t1)<<endl;
	return 0;
}
