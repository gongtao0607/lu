
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <cmath>
#include <cstdlib>
#ifdef __APPLE__
#include "pthread_barrier_osx.h"
#endif
#ifdef __HIPCC__
#define CUDA_CALLABLE __host__ __device__
#define BARRIER() __syncthreads()
#else
#define CUDA_CALLABLE
#include <pthread.h>
#define BARRIER() pthread_barrier_wait(&barrier)
#endif 
#ifdef GRAPHITE
#include "carbon_user.h"
#endif
#if defined(_MSC_VER)
#define lrand48() rand()
#define srand48(x) srand(x)
#endif  
using namespace std;
int N,P,B,NB;
class Block{
public:
	float*p;
	int b;
	Block(){p=NULL;}
	Block(int _b){p=NULL;b=_b;}
	CUDA_CALLABLE float& at(int x,int y){
		return p[x*b+y];
	}
};
class Matrix{
public:
	Block*p;
	int b;
	int nb;
	Matrix(){p=NULL;}
	Matrix(int _n,int _b){p=NULL;nb=_n/_b;b=_b;}
	CUDA_CALLABLE Block& getBlock(int x,int y){
		return p[x*nb+y];
	}
	CUDA_CALLABLE float& at(int x,int y){
		return getBlock(x/b,y/b).at(x%b,y%b);
	}
	void print(){
		for(int i=0;i<N;++i){
			for(int j=0;j<N;++j){
				cout<<at(i,j)<<" ";
			}
			cout<<endl;
		}
	}
}*A;
#ifndef __CUDACC__
pthread_barrier_t barrier;
#endif
void usage(char*n){
	cout<<"Usage: ./"<<n<<" N P B matrix_file\n\tN:NxN matrix\n\tP:P threads\n\tB:BxB block"<<endl;
}

// block b, referce block r, line "i" in b, line "k" in r, start column j, alpha
CUDA_CALLABLE void daxpy(Block&b , Block&r, int i, int k, int j, float alpha)
{
	for (int p = j; p<b.b; p++)     b.at(i,p) += alpha*r.at(k,p);
}
CUDA_CALLABLE void top_left(Block&b)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		/* modify subsequent columns */
		for (int i=k+1; i<b.b; i++) {
			b.at(i,k)/= b.at(k,k);
			alpha = -b.at(i,k);
			//length = n-k-1;
			daxpy(b, b, i, k, k+1, alpha);
		}
	}
}


CUDA_CALLABLE void top_right(Block&b, Block&r)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		for (int i=k+1; i<b.b; i++) {
			alpha = -r.at(i,k);
			daxpy(b, b, i, k, 0, alpha);
		}
	}
}


CUDA_CALLABLE void bottom_left(Block &b, Block&r)
{
	float alpha;
	for (int k=0; k<b.b; k++)
		for (int i=0; i<b.b; i++) {
			b.at(i,k) /= r.at(k,k);
			alpha = -b.at(i,k);
			daxpy(b, r, i, k, k+1, alpha);
		}
}


CUDA_CALLABLE void bottom_right(Block &b, Block&r1, Block&r2)
{
	float alpha;
	for (int k=0; k<b.b; k++) {
		for (int i=0; i<b.b; i++) {
			alpha = -r1.at(i,k);
			daxpy(b,r2,i,k,0,alpha);
		}
	}
}

#ifdef __HIPCC__
__global__ void thread_main(Matrix*A,int P)
#else
void* thread_main(void*p)
#endif
{

#ifdef __HIPCC__
	int thread_id=threadIdx.x;
#else
	int thread_id=*(int*)p;
	
#endif
	for(int round=0;round<A->nb;++round){
		//step 1, calculate top left
		if(thread_id==0){
			//A->getBlock(round,round);
			top_left(A->getBlock(round,round));
		}
		BARRIER();
		//step 2, bottom left and top right
		//x 0 2
		//1 x x
		//3 x x
		//0,1,2,3->()
		for(int i=thread_id;i<(A->nb-round-1)*2;i+=P){
			if(i&1){
				//bottom left
				//A->getBlock(round+(i>>1)+1,round);
				bottom_left(A->getBlock(round+(i>>1)+1,round),A->getBlock(round,round));
			}else{
				//top right
				//A->getBlock(round,round+(i>>1)+1);
				top_right(A->getBlock(round,round+(i>>1)+1),A->getBlock(round,round));
			}
		}
		BARRIER();
		//step 3, bottom right
		//x x x
		//x 0 1
		//x 2 3
		for(int i=thread_id;i<(A->nb-round-1)*(A->nb-round-1);i+=P){
			//A->getBlock(i/(NB-round-1)+round+1,i%(NB-round-1)+round+1)
			bottom_right(A->getBlock(i/(A->nb-round-1)+round+1,i%(A->nb-round-1)+round+1),
				A->getBlock(i/(A->nb-round-1)+round+1,round),
				A->getBlock(round,i%(A->nb-round-1)+round+1));
		}
		BARRIER();
	}
#ifdef __HIPCC__
#else
	return NULL;
#endif
}
#ifdef __HIPCC__
void cuda_upload(Matrix*rm, Matrix*lm){
	Block*blocks=new Block[NB*NB];
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			float*p;
			hipMalloc((void **)&p, sizeof(float)*B*B);
			blocks[i*NB+j].b=B;
			blocks[i*NB+j].p=p;
			hipMemcpy(p, lm->getBlock(i,j).p, sizeof(float)*B*B, hipMemcpyHostToDevice);
		}
	}
	Block*d_blocks;
	hipMalloc((void **)&d_blocks, sizeof(Block)*NB*NB);
	hipMemcpy(d_blocks, blocks, sizeof(Block)*NB*NB, hipMemcpyHostToDevice);
	
	Matrix*m=new Matrix(N,B);
	m->p=d_blocks;
	hipMemcpy(rm, m, sizeof(Matrix), hipMemcpyHostToDevice);
	
	delete m;
	delete[]blocks;
}
void cuda_download(Matrix*lm, Matrix*rm){
	hipMemcpy(lm, rm, sizeof(Matrix), hipMemcpyDeviceToHost);
	
	Block*blocks=new Block[NB*NB];
	hipMemcpy(blocks, lm->p, sizeof(Block)*NB*NB, hipMemcpyDeviceToHost);
	hipFree(lm->p);
	
	lm->p=blocks;
	for(int i=0;i<NB;++i){
		for(int j=0;j<NB;++j){
			float*p=new float[B*B];
			hipMemcpy(p, lm->getBlock(i,j).p, sizeof(float)*B*B, hipMemcpyDeviceToHost);
			hipFree(lm->getBlock(i,j).p);
			lm->getBlock(i,j).p=p;
		}
	}
}
#endif
int main(int argc, char**argv)
{
	if(argc<=3){
		usage(argv[0]);
		return -1;
	}
	N=atoi(argv[1]);
	P=atoi(argv[2]);
	B=atoi(argv[3]);
	if(N==0||P==0||B==0){
		usage(argv[0]);
		return -1;
	}
	NB=N/B;
	A=new Matrix(N,B);
	A->p=new Block[NB*NB];
	int i,j;
	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			A->getBlock(i,j).b=B;
			A->getBlock(i,j).p=new float[B*B];
		}
	}
	srand48(1);
#define MAXRAND 32768.0
	for(i=0;i<N;++i){
		for(j=0;j<N;++j){
			A->at(i,j)=((double)lrand48())/MAXRAND;
			if(i==j){
				A->at(i,j)*=10;
			}
		}
	}
#ifdef __HIPCC__
	Matrix*d_A;
	hipMalloc((void **)&d_A, sizeof(Matrix));
	cuda_upload(d_A,A);

	thread_main<<<1,P>>>(d_A,P);

	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			delete[] A->getBlock(i,j).p;
		}
	}
	delete[]A->p;
	cuda_download(A,d_A);
#else
#ifdef GRAPHITE
	CarbonEnableModels();
#endif
	pthread_barrier_init(&barrier,NULL,P);
	int*thread_args=new int[P];
	pthread_t*thread_handle=new pthread_t[P];
	for(i=1;i<P;++i){
		thread_args[i]=i;
		pthread_create(&thread_handle[i],NULL,thread_main,&thread_args[i]);
	}
	thread_args[0]=0;
	thread_main(&thread_args[0]);
	for(i=1;i<P;++i){
		pthread_join(thread_handle[i],NULL);
	}
	delete[]thread_args;
	delete[]thread_handle;
	pthread_barrier_destroy(&barrier);
#ifdef GRAPHITE
	CarbonDisableModels();
#endif
#endif
#ifndef GRAPHITE
//	A->print();
#endif

	for(i=0;i<NB;++i){
		for(j=0;j<NB;++j){
			delete[]A->getBlock(i,j).p;
		}
	}
	delete[]A->p;
	delete A;
	return 0;
}
